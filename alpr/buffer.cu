#include "NvInfer.h"
#include "half.h"
#include <cassert>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iterator>
#include <memory>
#include <new>
#include <numeric>
#include <string>
#include <vector>

inline unsigned int getElementSize(nvinfer1::DataType t)
{
    switch (t)
    {
    case nvinfer1::DataType::kINT32: return 4;
    case nvinfer1::DataType::kFLOAT: return 4;
    case nvinfer1::DataType::kHALF: return 2;
    case nvinfer1::DataType::kINT8: return 1;
    }
    throw std::runtime_error("Invalid DataType.");
}

template <typename A, typename B>
inline A divUp(A x, B n)
{
    return (x + n - 1) / n;
}

inline int64_t volume(const nvinfer1::Dims& d)
{
    return std::accumulate(d.d, d.d + d.nbDims, 1, std::multiplies<int64_t>());
}

template <typename AllocFunc, typename FreeFunc>
class GenericBuffer
{
public:
    GenericBuffer(nvinfer1::DataType type = nvinfer1::DataType::kFLOAT)
        : mSize(0)
        , mCapacity(0)
        , mType(type)
        , mBuffer(nullptr)
    {
    }

    GenericBuffer(size_t size, nvinfer1::DataType type)
        : mSize(size)
        , mCapacity(size)
        , mType(type)
    {
        if (!allocFn(&mBuffer, this->nbBytes()))
        {
            throw std::bad_alloc();
        }
    }

    GenericBuffer(GenericBuffer&& buf)
        : mSize(buf.mSize)
        , mCapacity(buf.mCapacity)
        , mType(buf.mType)
        , mBuffer(buf.mBuffer)
    {
        buf.mSize = 0;
        buf.mCapacity = 0;
        buf.mType = nvinfer1::DataType::kFLOAT;
        buf.mBuffer = nullptr;
    }

    GenericBuffer& operator=(GenericBuffer&& buf)
    {
        if (this != &buf)
        {
            freeFn(mBuffer);
            mSize = buf.mSize;
            mCapacity = buf.mCapacity;
            mType = buf.mType;
            mBuffer = buf.mBuffer;
            // Reset buf.
            buf.mSize = 0;
            buf.mCapacity = 0;
            buf.mBuffer = nullptr;
        }
        return *this;
    }

    void* data()
    {
        return mBuffer;
    }

    const void* data() const
    {
        return mBuffer;
    }

    size_t size() const
    {
        return mSize;
    }

    size_t nbBytes() const
    {
        return this->size() * getElementSize(mType);
    }


    void resize(size_t newSize)
    {
        mSize = newSize;
        if (mCapacity < newSize)
        {
            freeFn(mBuffer);
            if (!allocFn(&mBuffer, this->nbBytes()))
            {
                throw std::bad_alloc{};
            }
            mCapacity = newSize;
        }
    }


    void resize(const nvinfer1::Dims& dims)
    {
        return this->resize(volume(dims));
    }

    ~GenericBuffer()
    {
        freeFn(mBuffer);
    }

private:
    size_t mSize{0}, mCapacity{0};
    nvinfer1::DataType mType;
    void* mBuffer;
    AllocFunc allocFn;
    FreeFunc freeFn;
};

class DeviceAllocator
{
public:
    bool operator()(void** ptr, size_t size) const
    {
        return hipMalloc(ptr, size) == hipSuccess;
    }
};

class DeviceFree
{
public:
    void operator()(void* ptr) const
    {
        hipFree(ptr);
    }
};

class HostAllocator
{
public:
    bool operator()(void** ptr, size_t size) const
    {
        *ptr = malloc(size);
        return *ptr != nullptr;
    }
};

class HostFree
{
public:
    void operator()(void* ptr) const
    {
        free(ptr);
    }
};

using DeviceBuffer = GenericBuffer<DeviceAllocator, DeviceFree>;
using HostBuffer = GenericBuffer<HostAllocator, HostFree>;

class ManagedBuffer
{
public:
    DeviceBuffer deviceBuffer;
    HostBuffer hostBuffer;
};

class BufferManager
{
public:
    static const size_t kINVALID_SIZE_VALUE = ~size_t(0);

    BufferManager(std::shared_ptr<nvinfer1::ICudaEngine> engine, const int& batchSize,
        const nvinfer1::IExecutionContext* context = nullptr)
        : mEngine(engine)
        , mBatchSize(batchSize)
    {
        for (int i = 0; i < mEngine->getNbBindings(); i++)
        {
            auto dims = context ? context->getBindingDimensions(i) : mEngine->getBindingDimensions(i);
            size_t vol = context ? 1 : static_cast<size_t>(mBatchSize);
            nvinfer1::DataType type = mEngine->getBindingDataType(i);
            int vecDim = mEngine->getBindingVectorizedDim(i);
            if (-1 != vecDim) // i.e., 0 != lgScalarsPerVector
            {
                int scalarsPerVec = mEngine->getBindingComponentsPerElement(i);
                dims.d[vecDim] = divUp(dims.d[vecDim], scalarsPerVec);
                vol *= scalarsPerVec;
            }
            vol *= volume(dims);
            std::unique_ptr<ManagedBuffer> manBuf{new ManagedBuffer()};
            manBuf->deviceBuffer = DeviceBuffer(vol, type);
            manBuf->hostBuffer = HostBuffer(vol, type);
            mDeviceBindings.emplace_back(manBuf->deviceBuffer.data());
            mManagedBuffers.emplace_back(std::move(manBuf));
        }
    }

    std::vector<void*>& getDeviceBindings()
    {
        return mDeviceBindings;
    }

    const std::vector<void*>& getDeviceBindings() const
    {
        return mDeviceBindings;
    }

    void* getDeviceBuffer(const std::string& tensorName) const
    {
        return getBuffer(false, tensorName);
    }


    void* getHostBuffer(const std::string& tensorName) const
    {
        return getBuffer(true, tensorName);
    }

    size_t size(const std::string& tensorName) const
    {
        int index = mEngine->getBindingIndex(tensorName.c_str());
        if (index == -1)
            return kINVALID_SIZE_VALUE;
        return mManagedBuffers[index]->hostBuffer.nbBytes();
    }

    void dumpBuffer(std::ostream& os, const std::string& tensorName)
    {
        int index = mEngine->getBindingIndex(tensorName.c_str());
        if (index == -1)
        {
            os << "Invalid tensor name" << std::endl;
            return;
        }
        void* buf = mManagedBuffers[index]->hostBuffer.data();
        size_t bufSize = mManagedBuffers[index]->hostBuffer.nbBytes();
        nvinfer1::Dims bufDims = mEngine->getBindingDimensions(index);
        size_t rowCount = static_cast<size_t>(bufDims.nbDims >= 1 ? bufDims.d[bufDims.nbDims - 1] : mBatchSize);

        os << "[" << mBatchSize;
        for (int i = 0; i < bufDims.nbDims; i++)
            os << ", " << bufDims.d[i];
        os << "]" << std::endl;
        switch (mEngine->getBindingDataType(index))
        {
        case nvinfer1::DataType::kINT32: print<int32_t>(os, buf, bufSize, rowCount); break;
        case nvinfer1::DataType::kFLOAT: print<float>(os, buf, bufSize, rowCount); break;
        case nvinfer1::DataType::kHALF: print<half_float::half>(os, buf, bufSize, rowCount); break;
        case nvinfer1::DataType::kINT8: assert(0 && "Int8 network-level input and output is not supported"); break;
        }
    }

    template <typename T>
    void print(std::ostream& os, void* buf, size_t bufSize, size_t rowCount)
    {
        assert(rowCount != 0);
        assert(bufSize % sizeof(T) == 0);
        T* typedBuf = static_cast<T*>(buf);
        size_t numItems = bufSize / sizeof(T);
        for (int i = 0; i < static_cast<int>(numItems); i++)
        {
            if (rowCount == 1 && i != static_cast<int>(numItems) - 1)
                os << typedBuf[i] << std::endl;
            else if (rowCount == 1)
                os << typedBuf[i];
            else if (i % rowCount == 0)
                os << typedBuf[i];
            else if (i % rowCount == rowCount - 1)
                os << " " << typedBuf[i] << std::endl;
            else
                os << " " << typedBuf[i];
        }
    }

    void copyInputToDevice()
    {
        memcpyBuffers(true, false, false);
    }

    void copyOutputToHost()
    {
        memcpyBuffers(false, true, false);
    }

    void copyInputToDeviceAsync(const hipStream_t& stream = 0)
    {
        memcpyBuffers(true, false, true, stream);
    }

    void copyOutputToHostAsync(const hipStream_t& stream = 0)
    {
        memcpyBuffers(false, true, true, stream);
    }

    ~BufferManager() = default;

private:
    void* getBuffer(const bool isHost, const std::string& tensorName) const
    {
        int index = mEngine->getBindingIndex(tensorName.c_str());
        if (index == -1)
            return nullptr;
        return (isHost ? mManagedBuffers[index]->hostBuffer.data() : mManagedBuffers[index]->deviceBuffer.data());
    }

    void memcpyBuffers(const bool copyInput, const bool deviceToHost, const bool async, const hipStream_t& stream = 0)
    {
        for (int i = 0; i < mEngine->getNbBindings(); i++)
        {
            void* dstPtr
                = deviceToHost ? mManagedBuffers[i]->hostBuffer.data() : mManagedBuffers[i]->deviceBuffer.data();
            const void* srcPtr
                = deviceToHost ? mManagedBuffers[i]->deviceBuffer.data() : mManagedBuffers[i]->hostBuffer.data();
            const size_t byteSize = mManagedBuffers[i]->hostBuffer.nbBytes();
            const hipMemcpyKind memcpyType = deviceToHost ? hipMemcpyDeviceToHost : hipMemcpyHostToDevice;
            if ((copyInput && mEngine->bindingIsInput(i)) || (!copyInput && !mEngine->bindingIsInput(i)))
            {
                if (async)
                    hipMemcpyAsync(dstPtr, srcPtr, byteSize, memcpyType, stream);
                else
                    hipMemcpy(dstPtr, srcPtr, byteSize, memcpyType);
            }
        }
    }

    std::shared_ptr<nvinfer1::ICudaEngine> mEngine;
    int mBatchSize;
    std::vector<std::unique_ptr<ManagedBuffer>> mManagedBuffers;
    std::vector<void*> mDeviceBindings;
};
